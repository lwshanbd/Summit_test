#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
	hipError_t cuErr = call;                                                             \
	if(hipSuccess != cuErr){                                                             \
		printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
		exit(0);                                                                            \
	}                                                                                     \
}while(0)

int main()
{
    int NumOfDevices;

    cudaErrorCheck( hipGetDeviceCount(&NumOfDevices));

    printf("number of devices is %d\n", NumOfDevices);

    return 0;

}